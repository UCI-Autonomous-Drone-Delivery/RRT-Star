#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <vector>
#include <time.h>
#include <cmath>
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>

#include "Graph.hpp"
using namespace std;

__global__ void checkIfNeighbor(float r,int n,float x, float y,float z, float* xArr,float* yArr,float* zArr,bool* output) {
	int i=threadIdx.x;
	if(i<n) {

		if(5<=r) {
			output[i]=true;
		}
		else {
			output[i]=false;
		}
	}

}

__global__
void initializeArr(float* x,float* y,float* z,Node* node) {

}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


vector<Node*> nearestNeighborsGPU(Node* node, float r,int n) {
	Coord* coord=node->coord;

	//arrays of info to pass to gpu
	float *xArr;
	float *yArr;
	float *zArr;
	float x,y,z;
	bool *output;
	x=coord->x;
	y=coord->y;
	z=coord->z;
	gpuErrchk(hipMallocManaged(&xArr, n*sizeof(float)));
	hipMallocManaged(&yArr, n*sizeof(float));
	hipMallocManaged(&zArr, n*sizeof(float));
	//hipMallocManaged(&output, n*sizeof(bool));

	for(int i=0;i<n;i++) {
		cout<< "made it here\n";
		xArr[i]=5.0f;//adj_list.at(i)->coord->x;
		yArr[i]=5.0f;//adj_list.at(i)->coord->y;
		zArr[i]=5.0f;//adj_list.at(i)->coord->z;
	}

	//checkIfNeighbor<<<1,256>>>(r,num_nodes,x,y,z,xArr,yArr,zArr,output);
	hipDeviceSynchronize();
	vector<Node*> neighbors;
	
	return neighbors;
}

int main() {
	Coord* startingCoord=new Coord(60,10,29);

	//Graph graph(NUMNODES,startingCoord);

	Coord* c=new Coord(50,50,50);
	Node* n=new Node(-1,c);

	auto t1 = Clock::now();

	vector<Node*> neighbors=nearestNeighborsGPU(n,50,256);
    auto t2 = Clock::now();

	cout << "Delta t2-t1: " 
    	 << std::chrono::duration_cast<chrono::nanoseconds>(t2 - t1).count()
    	 << " nanoseconds" <<endl;

	cin.get();

}